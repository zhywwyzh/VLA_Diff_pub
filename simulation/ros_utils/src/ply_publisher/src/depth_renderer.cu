#include "hip/hip_runtime.h"
#include "ply_publisher/depth_renderer.h"
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <cmath>
#include <limits>

using namespace depth_renderer;

static __device__ __forceinline__ float3 mul3x3v(const float R[9], const float3& v) {
  return make_float3(
    R[0]*v.x + R[1]*v.y + R[2]*v.z,
    R[3]*v.x + R[4]*v.y + R[5]*v.z,
    R[6]*v.x + R[7]*v.y + R[8]*v.z
  );
}

static __device__ __forceinline__ void atomicMinFloat(float* addr, float val) {
  unsigned int* uaddr = reinterpret_cast<unsigned int*>(addr);
  unsigned int  old   = atomicAdd(uaddr, 0u);
  while (__uint_as_float(old) > val) {
    unsigned int assumed = old;
    old = atomicCAS(uaddr, assumed, __float_as_uint(val));
    if (old == assumed) break;
  }
}

__global__ void projectAndZMinKernel(
  const float* __restrict__ pts_xyz, int n,
  float* __restrict__ depth, int W, int H,
  float fx, float fy, float cx, float cy,
  float depth_min, float depth_max,
  float t_wb_x, float t_wb_y, float t_wb_z,
  const float* __restrict__ R_ow,
  int r)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n) return;

  float wx = pts_xyz[3*i + 0];
  float wy = pts_xyz[3*i + 1];
  float wz = pts_xyz[3*i + 2];

  float dx = wx - t_wb_x;
  float dy = wy - t_wb_y;
  float dz = wz - t_wb_z;

  float3 pc = mul3x3v(R_ow, make_float3(dx,dy,dz));
  float Z = pc.z;
  if (Z <= depth_min || Z > depth_max) return;

  float uf = fx * pc.x / Z + cx;
  float vf = fy * pc.y / Z + cy;
  int u0 = __float2int_rn(uf);
  int v0 = __float2int_rn(vf);
  if (u0 < 0 || u0 >= W || v0 < 0 || v0 >= H) return;

  int umin = max(0, u0 - r);
  int umax = min(W-1, u0 + r);
  int vmin = max(0, v0 - r);
  int vmax = min(H-1, v0 + r);

  for (int v = vmin; v <= vmax; ++v) {
    int base = v * W;
    for (int u = umin; u <= umax; ++u) {
      atomicMinFloat(&depth[base + u], Z);
    }
  }
}

static inline void gpuCheck(hipError_t e, const char* msg) {
  if (e != hipSuccess) {
    throw std::runtime_error(std::string("CUDA Error: ") + msg + " - " + hipGetErrorString(e));
  }
}

bool depth_renderer::renderDepthCUDA(const float* pts_xyz, int n,
                                     const Pose& pose, const Intrinsics& K,
                                     float* depth_out)
{
  if (!pts_xyz || !depth_out || n <= 0) return false;

  const int W = K.W, H = K.H;
  const size_t depth_bytes = sizeof(float)*W*H;

  float *d_pts=nullptr, *d_depth=nullptr, *d_R=nullptr;
  try {
    gpuCheck(hipMalloc(&d_pts, sizeof(float)*3*n), "malloc d_pts");
    gpuCheck(hipMemcpy(d_pts, pts_xyz, sizeof(float)*3*n, hipMemcpyHostToDevice), "cpy pts");

    gpuCheck(hipMalloc(&d_depth, depth_bytes), "malloc d_depth");
    // 用 0x7f 填充通常得到 NaN/Inf，用于“极大值”；后面会把非命中转成 +INF
    gpuCheck(hipMemset(d_depth, 0x7f, depth_bytes), "memset depth");

    gpuCheck(hipMalloc(&d_R, sizeof(float)*9), "malloc d_R");
    gpuCheck(hipMemcpy(d_R, pose.R_ow, sizeof(float)*9, hipMemcpyHostToDevice), "cpy R");

    dim3 block(256);
    dim3 grid((n + block.x - 1)/block.x);
    projectAndZMinKernel<<<grid, block>>>(
      d_pts, n, d_depth, W, H,
      K.fx, K.fy, K.cx, K.cy,
      K.depth_min, K.depth_max,
      pose.t_wb[0], pose.t_wb[1], pose.t_wb[2],
      d_R, K.splat_radius_px
    );
    gpuCheck(hipPeekAtLastError(), "kernel launch");
    gpuCheck(hipDeviceSynchronize(), "kernel sync");

    gpuCheck(hipMemcpy(depth_out, d_depth, depth_bytes, hipMemcpyDeviceToHost), "cpy back");

    // 归一未命中像素
    for (int i=0;i<W*H;++i) {
      float& z = depth_out[i];
      if (!(z > 0.f) || !std::isfinite(z)) z = std::numeric_limits<float>::infinity();
    }

    hipFree(d_pts); hipFree(d_depth); hipFree(d_R);
    return true;
  } catch (...) {
    if (d_pts)   hipFree(d_pts);
    if (d_depth) hipFree(d_depth);
    if (d_R)     hipFree(d_R);
    return false;
  }
}
